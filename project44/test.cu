#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define N 10000000
#define THREADS_PER_BLOCK 1024

__global__ void vector_add(float *out, float *a, float *b, int n) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride) {
        out[i] = a[i] + b[i];
    }
}

int main() {
    clock_t start, end;
    double cpuTimeUsed;

    float *h_a, *h_b, *h_out;  // Host pointers
    float *d_a, *d_b, *d_out;  // Device pointers

    size_t size = sizeof(float) * N;

    // Allocate host memory
    h_a = (float *)malloc(size);
    h_b = (float *)malloc(size);
    h_out = (float *)malloc(size);

    // Initialize host arrays
    for (int i = 0; i < N; i++) {
        h_a[i] = 1.0f;
        h_b[i] = 2.0f;
    }

    // Allocate device memory
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_out, size);

    // Copy data from host to device
    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

    // Start timing
    start = clock();

    // Launch kernel
    int blocks = (N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    vector_add<<<blocks, THREADS_PER_BLOCK>>>(d_out, d_a, d_b, N);

    // Copy result back to host
    hipMemcpy(h_out, d_out, size, hipMemcpyDeviceToHost);

    // Stop timing
    end = clock();

    cpuTimeUsed = ((double)(end - start)) / CLOCKS_PER_SEC;
    printf("Time taken: %f seconds\n", cpuTimeUsed);

    // Free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);

    // Free host memory
    free(h_a);
    free(h_b);
    free(h_out);

    return 0;
}
